#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>
#include <limits.h>
#include <assert.h>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include "utils.h"
#include "tsp_solve.h"
#include "gen_city.h"

#define t_num 1024
#define GRID_SIZE 1024
#define LINE_BUF_LEN 100

/*************************************************** 
* Desc: Simulated Annealing for roundtrips
* Notes:
*  For more samples define GRID_SIZE as a multiple of t_num such as 512000, 2048000, or the (max - 1024) grid size 2147482623
*  Some compliation options that can speed things up
*  --use_fast_math 
* --optimize=5
*  --gpu-architecture=compute_35
*  I use something like
*   nvcc --optimize=5 --use_fast_math -arch=compute_35 tsp_cuda.cu -o tsp_cuda
*
*  We are going to try some stuff for temp from this adaptive simulated annealing paper
*  https://arxiv.org/pdf/cs/0001018.pdf
****************************************************/

/***************************************************************************
*   Variable Descriptions:
* tsp_name: [const char()]
*  - The name of the tsp file we get our meta data and location data from
* N, N_g: [integer(1)]
*  - The number of cities. We also place this on the device for checking corner cases
* i,j,m [integer(1)]
*  - Standard iters
* salesman_route [unsigned integer(N + 1)]
*  - The route that we will traverse over, starts and ends at the same location
* city_swap_one_h/g: [integer(t_num)]
*  - Host/Device memory for city one
* city_swap_two_h/g: [integer(t_num)]
*  - Host/Device memory for city two
* flag_h/g: [integer(t_num)]
*  - Host/Device memory for flag of accepted step
* salesman_route_g: [integer(N)]
*  - Device memory for the salesmans route
* starting loss [float(1)]
*  - The loss from the naive route
* T,T_start,T_g [float(1)]
*  - T is the temperature that is changed as we iterate through SA, T_start is the initial temperature
*  - T_g is device memory for the temperature within SA
* r_h/g:  [float(t_num)]
*  - host/device memory for the random number when deciding acceptance
* flag_h/g: [integer(t_num)]
*  - host/device memory for acceptance vector
* original_loss_h/g: [integer(1)]
*  - The host/device current loss function
* new_loss_h/g: [integer(t_num)]
*  - The host/device memory for the proposal loss function
* d_state [struct hiprandState(1)]
*  - The beginning state of the random number generation
* 
*****************************************************************************/
 int main(){
 
     // Set up the structs, location is allocated in read_tsp
     // meta and location come from utils.h
     const char *tsp_name = "dsj1000.tsp";
     read_tsp(tsp_name);
     unsigned int N = meta -> dim, *N_g;     
     // start counters for cities
     unsigned int i;
     unsigned int *salesman_route = (unsigned int *)malloc((N + 1) * sizeof(unsigned int));
     float original_loss = 0;
     float starting_loss = original_loss;
     // SET TEMP HERE
     float T_start = 5, *T_g, *T_start_g, *r_g;
     float T = T_start;
     float *r_h = (float *)malloc(GRID_SIZE * sizeof(float));
     float new_loss_h = 0;
     float iter = 1.0;
     unsigned int *city_swap_one_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
     unsigned int *city_swap_two_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
     unsigned int *flag_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
     unsigned int *city_swap_one_g, *city_swap_two_g, *salesman_route_g, *flag_g;
     unsigned int global_flag_h = 0, *global_flag_g;
     coordinates *location_g;
     hiprandState_t* states;
     // Create space on the device for each var
     hipError_t err = hipMalloc((void**)&city_swap_one_g, GRID_SIZE * sizeof(unsigned int));
     //printf("\n Cuda malloc city swap one: %s \n", hipGetErrorString(err));
     hipMalloc((void**)&city_swap_two_g, GRID_SIZE * sizeof(unsigned int));
     hipMalloc((void**)&location_g, N * sizeof(coordinates));
     hipMalloc((void**)&salesman_route_g, (N + 1) * sizeof(unsigned int));
     hipMalloc((void**)&T_g, sizeof(float));
     hipMalloc((void**)&r_g, GRID_SIZE * sizeof(float));
     hipMalloc((void**)&flag_g, GRID_SIZE * sizeof(unsigned int));
     hipMalloc((void**)&global_flag_g, sizeof(unsigned int));
     hipMalloc((void**)&N_g, sizeof(unsigned int));
     hipMalloc((void**)&T_start_g, sizeof(float));
     // Make space for the state of the RNG
     hipMalloc((void**) &states, N * sizeof(hiprandState_t));
     // Copy the city locations to device as well as the number of cities
     hipMemcpy(location_g, location, N * sizeof(coordinates), hipMemcpyHostToDevice);
     hipMemcpy(N_g, &N, sizeof(unsigned int), hipMemcpyHostToDevice);
     hipMemcpy(T_start_g, &T_start, sizeof(float), hipMemcpyHostToDevice);
     

     
     // Set up the RNG
    
     

     // just make one inital guess route, a simple linear path
     for (i = 0; i <= N; i++)
         salesman_route[i] = i;
         
     // Set the starting and end points to be the same
     salesman_route[N] = salesman_route[0];
    

     // Calculate the original loss
     for (i = 0; i < N; i++){
         original_loss += (location[salesman_route[i]].x - location[salesman_route[i+1]].x) *
                          (location[salesman_route[i]].x - location[salesman_route[i+1]].x) +
                          (location[salesman_route[i]].y - location[salesman_route[i+1]].y) *
                          (location[salesman_route[i]].y - location[salesman_route[i+1]].y);
     }
     starting_loss = original_loss;
     printf("Original Loss is: %.6f \n", original_loss);
     printf("Number of cities: %d \n", N); 
     //Best for 100,000: The starting loss was 33,346,203,648 and the final loss was 10,243,860,480 
     while (T > 1){
         hipMemcpy(T_g, &T, sizeof(float), hipMemcpyHostToDevice);
         // Init parameters
         init<<<GRID_SIZE, 1>>>(time(0), states);

        
        
         // Number of thread blocks in grid
         dim3 blocksPerGrid(GRID_SIZE/t_num,1,1);
         dim3 threadsPerBlock(t_num,1,1);
    
         genCity<<<GRID_SIZE, 1>>>(states, city_swap_one_g,city_swap_two_g, N_g,
                            T_start_g, T_g, flag_g, global_flag_g, r_g);
         hipDeviceSynchronize();        
         //hipMemcpy(city_swap_two_h, city_swap_two_g, GRID_SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
         //for (int k = 0; k < GRID_SIZE; k++)
         //  printf("city two is: %d \n", city_swap_two_h[k]);        
         tsp<<<blocksPerGrid, threadsPerBlock, 0>>>(city_swap_one_g, city_swap_two_g,
                                                    location_g, salesman_route_g,
                                                    T_g, r_g, flag_g, global_flag_g,
                                                    N_g);

         hipDeviceSynchronize();
         hipMemcpy(&global_flag_h, global_flag_g, sizeof(unsigned int), hipMemcpyDeviceToHost);
         if (global_flag_h != 0){          
          hipMemcpy(flag_h, flag_g, GRID_SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
          /* 
           Here we check for a success
            The first proposal trip accepted becomes the new starting trip 
          */
           for (i = 0; i < GRID_SIZE; i++){
             if (flag_h[i] == 0){
               //printf("Original Loss: %.6f \n", original_loss);
               //printf("Proposed Loss: %.6f \n", new_loss_h[i]);
               continue;
             } else {
               // switch the two cities that led to an accepted proposal
                 unsigned int tmp = salesman_route[city_swap_one_h[i]];
                 salesman_route[city_swap_one_h[i]] = salesman_route[city_swap_two_h[i]];
                 salesman_route[city_swap_two_h[i]] = tmp;
                 if (tmp == 0)
                 salesman_route[N] = tmp;
                 new_loss_h = 0;
                 for (i = 0; i < N - 1; i++){
                   new_loss_h += (location[salesman_route[i]].x - location[salesman_route[i+1]].x) *
                                 (location[salesman_route[i]].x - location[salesman_route[i+1]].x) +
                                 (location[salesman_route[i]].y - location[salesman_route[i+1]].y) *
                                 (location[salesman_route[i]].y - location[salesman_route[i+1]].y);
                 }

                 // set old loss function to new
                 original_loss = new_loss_h;
                 //decrease temp
                    /*
                    printf("Best found trip so far\n");
                    for (j = 0; j < N; j++){
                       printf("%d ", salesman_route[j]);
                    }
                    */
               break;
             }
           }
         }
     
     //if ((int)iter % 1000 == 0){
         printf(" Current Temperature is %.6f \n", T);
         printf("\n Current Loss is: %.6f \n", original_loss);
         printf("\n Current Iteration is: %.6f \n", iter);
     //}
     //T = 1;
     T = T_start /log(iter);
     iter += 1.0f;
     }
     printf("The starting loss was %.6f and the final loss was %.6f \n", starting_loss, original_loss);
     /*
     printf("\n Final Route:\n");
     for (i = 0; i < N; i++)
       printf("%d ",salesman_route[i]);
     */    
     hipFree(city_swap_one_g);
     hipFree(city_swap_two_g);
     hipFree(location_g);
     hipFree(salesman_route_g);
     hipFree(T_g);
     hipFree(r_g);
     hipFree(flag_g);
     hipFree(N_g);
     hipFree(T_start_g);
     free(salesman_route);
     free(city_swap_one_h);
     free(city_swap_two_h);
     free(flag_h);
     free(location);
     free(meta);
     return 0;
}
             
         
         
         

