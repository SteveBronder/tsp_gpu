#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<string.h>
#include<limits.h>
#include<assert.h>
#include<iostream>
#include<fstream>
#include<ctype.h>

// If NDEBUG is defined, cudaCheckError() will be empty 
//#define NDEBUG
#include "kernels/utils.h"
#include "kernels/initialize_rng.h"
#include "kernels/swap_sampler.h"
#include "kernels/insert_sampler.h"
#include "kernels/opt2_sampler.h"


#define t_num 1024
#define GRID_SIZE 131072


/*
For more samples define GRID_SIZE as a multiple of t_num such as 512000, 2048000, or the (max - 1024) grid size 2147482623
A good grid size is the number of SM's you have times the number of blocks each can take in times max threads per block
I have 8 cores that can hold 16 blocks of 1024 cores so my best is 131072
Some compiler options that can speed things up
--use_fast_math
--optimize=5
--gpu-architecture=compute_35
I use something like
NOTE: You need to use the -lcurand flag to compile for the RNG.
nvcc --optimize=5 --use_fast_math -arch=compute_35 main_solver.cu -o tsp_cuda -lcurand
*/

int main(int argc, char *argv[]){

// Reading in inputs
    if (argc == 1){
        printf("Inputs: \n" 
               "(Required) input_file.tsp: [char()] \n"
               " - The name of the tsp file, excluding .tsp at the end, containing the cities to travel over. \n"
               "(Optional) -trip: [char()] \n"
               " - The name of the csv file, excluding .csv, containing a previously found trip."
               " If missing, a linear route is generated as the starting trip. \n"
               "(Optional) -temp: [float(1)] \n" 
               " - The initial starting temperature. Default is 1000 \n"
               "(Optional) -decay: [float(1)]  \n"
               " - The decay rate for the annealing schedule. Default is .99 \n"
               "(Optional) -maxiter: [integer(1)]  \n"
               " - The maximum number of iterations until failure. \n"
               "  Default is -1, which runs until temperature goes to the minimum.\n"
               "(Optional) -global_search: [float(1)]  \n"
               " - A parameter that controls the variance of the second city search space,\n"
               "   such that the variance is [30 + exp(global_search/Temp) * N]. default is .01.\n"
               "  See An example of what this controls here:\n"
               "(Optional) -local_search: [float(1)]  \n"
               " - A parameter that controls the variance of the second city search space,\n"
               "   such that the variance is [30 + exp(local_search/Temp) * N]. default is 1.\n");
               
        return 1;
    }
    
	const char *tsp_name = concat(argv[1], ".tsp");
	coordinates *location_g;
	read_tsp(tsp_name);
    unsigned int N = meta->dim, *N_g;
	unsigned int i;
    unsigned int *salesman_route = (unsigned int *)malloc((N + 1) * sizeof(unsigned int));
    float sample_area_local, sample_area_global, *sample_area_local_g, *sample_area_global_g;
    sample_area_global = 0.01;
    sample_area_local = 1;
	// just make one inital guess route, a simple linear path
	for (i = 0; i <= N; i++)
		salesman_route[i] = i;

	// Set the starting and end points to be the same
	salesman_route[N] = salesman_route[0];
    
    // Get loss
    float T[1], *T_g;
	T[1] = .03;
	float decay = 0.99;
	int maxiter = -1;
    // Get starting trip
	for (i = 0; i <= N; i++)
	    salesman_route[i] = i;
	// Set the starting and end points to be the same
    salesman_route[N] = salesman_route[0];
    
    
    // read in options
    for (int i = 1; i < argc; i++) {
        if (i + 1 != argc)    {
            if (strcmp(argv[i], "-trip=") == 0) {          
                const char *trip_name = concat(argv[i + 1], ".csv");
                read_trip(trip_name, salesman_route);    
            }
            if (strcmp(argv[i], "-temp=") == 0) {           
                // If atof cannot convert to a float, it returns 0
	            float user_temp = atof(argv[i + 1]);
	            if ( user_temp == 0){
	                printf("Error: Initial Temperature must be a non-zero number\n");
	                return 1;
	            }
	            T[0] = user_temp;
	            T[1] = T[0];
	        }
	        if (strcmp(argv[i], "-maxiter=") == 0) {           
                // If atof cannot convert to a float, it returns 0
	            float user_iter = atoi(argv[i + 1]);
	            if ( user_iter == 0){
	                printf("Error: max iter cannot be zero\n");
	                return 1;
	            }
	            maxiter = user_iter;
	        }
            if (strcmp(argv[i], "-decay=") == 0) {           
                // If atoi cannot convert to number, it returns 0
                float user_decay = atof(argv[i + 1]);
	            if (user_decay == 0){
	                printf("Error: Decay must be a number from 0 to 1\n");
	                return 1;
	            } else if (user_decay >= 1 || user_decay <= 0){
	                printf("Error: Decay must be a number from 0 to 1\n");
	                return 1;
	            } else {
	               decay = user_decay;
	            }    
            }
            if (strcmp(argv[i], "-global_search=") == 0) {           
                // If atoi cannot convert to number, it returns 0
                float user_global = atof(argv[i + 1]);
	            if (user_global == 0){
	                printf("Error: global search param must be greater than 0. \n");
	                return 1;
	            } else {
	               sample_area_global = user_global;
	            }    
            }
            if (strcmp(argv[i], "-local_search=") == 0) {           
                // If atoi cannot convert to number, it returns 0
                float user_local = atof(argv[i + 1]);
	            if (user_local == 0){
	                printf("Error: local search param must be greater than 0. \n");
	                return 1;
	            } else {
	               sample_area_local = user_local;
	            }    
            }
        }
    }

	// Calculate the original loss 
	float original_loss = 0;
	for (i = 0; i < N; i++){
		original_loss += sqrtf((location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
			(location[salesman_route[i]].x - location[salesman_route[i + 1]].x) +
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y));
	}
	printf("Original Loss is:  %0.6f \n", original_loss);
	float optimized_loss_restart = original_loss;
	// Keep the original loss for comparison pre/post algorithm
	// SET THE LOSS HERE
	
	/*
	Defining device variables:
	city_swap_one_h/g: [integer(t_num)]
	- Host/Device memory for city one
	city_swap_two_h/g: [integer(t_num)]
	- Host/Device memory for city two
	flag_h/g: [integer(t_num)]
	- Host/Device memory for flag of accepted step
	salesman_route_g: [integer(N)]
	- Device memory for the salesmans route
	flag_h/g: [integer(t_num)]
	- host/device memory for acceptance vector
	original_loss_g: [integer(1)]
	- The device memory for the current loss function
	(DEPRECATED)new_loss_h/g: [integer(t_num)]
	- The host/device memory for the proposal loss function
	*/
	unsigned int *city_swap_one_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
	unsigned int *city_swap_two_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
	unsigned int *flag_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
	unsigned int *salesman_route_g, *salesman_route_2g, *salesman_route_restartg, *flag_g, *city_swap_one_g, *city_swap_two_g;
	int global_flag_h = 0, *global_flag_g_1, *global_flag_g_2, *global_flag_g_3;
	int *global_flag_g_4, *global_flag_g_5;

    
	hipMalloc((void**)&city_swap_one_g, GRID_SIZE * sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&city_swap_two_g, GRID_SIZE * sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&location_g, N * sizeof(coordinates)); 
	cudaCheckError();
	hipMalloc((void**)&salesman_route_g, (N + 1) * sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&salesman_route_2g, (N + 1) * sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&salesman_route_restartg, (N + 1) * sizeof(unsigned int));
    cudaCheckError();
	hipMalloc((void**)&T_g, sizeof(float));
	cudaCheckError();
	hipMalloc((void**)&sample_area_global_g, sizeof(float));
	cudaCheckError();
    hipMalloc((void**)&sample_area_local_g,  sizeof(float));
	cudaCheckError();
	hipMalloc((void**)&flag_g, GRID_SIZE * sizeof(int));
	cudaCheckError();
	hipMalloc((void**)&global_flag_g_1, sizeof(int));
	cudaCheckError();
	hipMalloc((void**)&global_flag_g_2, sizeof(int));
	cudaCheckError();
	hipMalloc((void**)&global_flag_g_3, sizeof(int));
	cudaCheckError();
	hipMalloc((void**)&global_flag_g_4, sizeof(int));
	cudaCheckError();
	hipMalloc((void**)&global_flag_g_5, sizeof(int));
	cudaCheckError();
	hipMalloc((void**)&N_g, sizeof(unsigned int));
	cudaCheckError();


	hipMemcpy(location_g, location, N * sizeof(coordinates), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(salesman_route_g, salesman_route, (N + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(salesman_route_2g, salesman_route, (N + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(salesman_route_restartg, salesman_route, (N + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    cudaCheckError();
	hipMemcpy(global_flag_g_1, &global_flag_h, sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(global_flag_g_2, &global_flag_h, sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(global_flag_g_3, &global_flag_h, sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(global_flag_g_4, &global_flag_h, sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(global_flag_g_5, &global_flag_h, sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(N_g, &N, sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(sample_area_global_g, &sample_area_global, sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(sample_area_local_g, &sample_area_local,  sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaCheckError();
	// Beta is the decay rate
	//float beta = 0.0001;
	// We are going to try some stuff for temp from this adaptive simulated annealing paper
	// https://arxiv.org/pdf/cs/0001018.pdf

	// Number of thread blocks in grid
	// X is for the sampling, y is for manipulating the salesman's route
	dim3 blocksPerSampleGrid(GRID_SIZE / t_num, 1, 1);
	dim3 blocksPerTripGrid((N / t_num) + 1, 1, 1);
	dim3 threadsPerBlock(t_num, 1, 1);

	// Trying out random gen in cuda
	hiprandState_t* states;

	/* allocate space on the GPU for the random states */
	hipMalloc((void**)&states, GRID_SIZE * sizeof(hiprandState_t));
	init << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(time(0), states);

	//time counter
	time_t t_start, t_end;
	t_start = time(NULL); 
    long int iter = 1;
    int sames = 0;
	while (T[0] > 0.01 / log(20 * N))
	{
		// Copy memory from host to device
		hipMemcpy(T_g, T, sizeof(float), hipMemcpyHostToDevice);
		i = 1;              
		 
		while (i<2000){                                                                                         // key

	
			swapStep << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				location_g, salesman_route_g,
				T_g, global_flag_g_1, N_g,
				states, sample_area_global_g); 
			cudaCheckError();
			
			swapUpdate << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				salesman_route_g, global_flag_g_1);
			cudaCheckError();
			
			swapStep << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				location_g, salesman_route_g,
				T_g, global_flag_g_2, N_g,
				states,sample_area_local_g);
			cudaCheckError();
			
			swapUpdate << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				salesman_route_g, global_flag_g_2); 
			cudaCheckError(); 
		
			twoOptStep << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				location_g, salesman_route_g, 
				T_g, global_flag_g_3, N_g,
				states, sample_area_global_g);
			cudaCheckError(); 
			
			insertionUpdateTrip << <blocksPerTripGrid, threadsPerBlock, 0 >> >(salesman_route_g, salesman_route_2g, N_g);
			cudaCheckError();
			
			opt2Update << <blocksPerTripGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				salesman_route_g, salesman_route_2g, global_flag_g_3);
			cudaCheckError();
			
			twoOptStep << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				location_g, salesman_route_g,
				T_g, global_flag_g_3, N_g,
				states, sample_area_local_g);
			cudaCheckError();
			
			insertionUpdateTrip << <blocksPerTripGrid, threadsPerBlock, 0 >> >(salesman_route_g, salesman_route_2g, N_g);
			cudaCheckError();
			
			opt2Update << <blocksPerTripGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				salesman_route_g, salesman_route_2g, global_flag_g_3);
			cudaCheckError();

			insertionStep << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				location_g, salesman_route_g,
				T_g, global_flag_g_4, N_g,
				states, sample_area_global_g);
			cudaCheckError();
			
			insertionUpdateTrip << <blocksPerTripGrid, threadsPerBlock, 0 >> >(salesman_route_g, salesman_route_2g, N_g);
			cudaCheckError();
			
			insertionUpdate2 << <blocksPerTripGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				salesman_route_g, salesman_route_2g, global_flag_g_4);
			cudaCheckError(); 
			
			insertionStep << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				location_g, salesman_route_g,
				T_g, global_flag_g_5, N_g,
				states, sample_area_local_g);
			cudaCheckError();
			
			insertionUpdateTrip << <blocksPerTripGrid, threadsPerBlock, 0 >> >(salesman_route_g, salesman_route_2g, N_g);
			cudaCheckError();
			
		    insertionUpdate2 << <blocksPerTripGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				salesman_route_g, salesman_route_2g, global_flag_g_5);
			cudaCheckError();

			i++;
		}
		hipMemcpy(salesman_route, salesman_route_g, (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
		cudaCheckError();
		float optimized_loss = 0;
		for (i = 0; i < N; i++){
			optimized_loss += sqrt((location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
				(location[salesman_route[i]].x - location[salesman_route[i + 1]].x) +
				(location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
				(location[salesman_route[i]].y - location[salesman_route[i + 1]].y));
		}
		printf("| Loss: %.6f | Temp: %f | Iter: %ld |\n", optimized_loss, T[0], iter);
		T[0] = T[0] * decay;
				iter++;
		// This grabs the best trip overall
		if (optimized_loss < optimized_loss_restart){
		    optimized_loss_restart = optimized_loss;
		    insertionUpdateTrip <<<blocksPerTripGrid, threadsPerBlock, 0 >>>(salesman_route_g, salesman_route_restartg, N_g);
			cudaCheckError();
			sames = 0;
	    } else if (abs(optimized_loss - optimized_loss_restart) < 2){
	    // If we are only gaining by one then we can start speeding things up
	        sames++;
	        if (sames > 10){
	            T[0] = T[0] * 0.8;
	            }
	    }
	    if (maxiter > 0 && maxiter < iter)
            break;

	}
	
	t_end = time(NULL);
	printf("time = %f\n", difftime(t_end, t_start));

	hipMemcpy(salesman_route, salesman_route_g, (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	cudaCheckError();
	
	// We have to redefine optimized loss for some reason?
	float optimized_loss = 0;
	for (i = 0; i < N; i++){
		optimized_loss += sqrt((location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
			(location[salesman_route[i]].x - location[salesman_route[i + 1]].x) +
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y));
	}
	
	// If it's worse than the restart make the route the restart.
	if (optimized_loss > optimized_loss_restart){
        hipMemcpy(salesman_route, salesman_route_restartg, (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
        cudaCheckError();
	}
	
	optimized_loss = 0;
	for (i = 0; i < N; i++){
		optimized_loss += sqrt((location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
			(location[salesman_route[i]].x - location[salesman_route[i + 1]].x) +
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y));
	}
	
	printf("Original Loss is:  %0.6f \n", original_loss); 
	printf("Optimized Loss is: %.6f \n", optimized_loss);

	// Write the best trip to CSV
	FILE *best_trip;
	const char *filename = concat(argv[1],"_trip.csv");
	best_trip = fopen(filename, "w+");
	fprintf(best_trip, "location,coordinate_x,coordinate_y\n");
	for (i = 0; i < N + 1; i++){
		fprintf(best_trip, "%d,%.6f,%.6f\n",
			salesman_route[i],
			location[salesman_route[i]].x,
			location[salesman_route[i]].y);
	}
	fclose(best_trip);

	hipFree(location_g);
	cudaCheckError();
	hipFree(salesman_route_g);
	cudaCheckError();
	hipFree(salesman_route_2g);
	cudaCheckError();
	hipFree(T_g);
	cudaCheckError();
	hipFree(flag_g);
	cudaCheckError();
	hipFree(salesman_route_restartg);
	cudaCheckError();
	hipFree(sample_area_global_g);
	cudaCheckError();
	hipFree(sample_area_local_g);
	cudaCheckError();
	free(salesman_route);
	free(city_swap_one_h);
	free(city_swap_two_h);
	free(flag_h);
	free(location);
	getchar();
	return 0;
}
