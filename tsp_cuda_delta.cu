#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>


#define N 15000 
#define t_num 1024
#define GRID_SIZE 512000
 
 /* 
 Some compliation options that can speed things up
 --use_fast_math 
 --optimize=5
 --gpu-architecture=compute_35
 I use something like
  nvcc --optimize=5 --use_fast_math -arch=compute_35 tsp_cuda.cu -o tsp_cuda
 */
 
 /* TSP With Only Difference Calculation
Input:
- i: A vector of cities to swap for the first swap choice
- k: A vector of cities to swap for the second swap choice
- dist: The distance matrix of each city
- salesman_route: The route the salesman will travel
- T: The current temperature
- r: The random number to compare against for S.A.
*/
__global__ static void tsp(unsigned int* city_one,unsigned int* city_two,
                           float *dist, unsigned int *salesman_route,
                           float *T, float *r,
                           unsigned int *flag){
    
    const int tid = threadIdx.x;
    float delta, p, b = 1;
    
    // first city to swap
    int salesman_route_city_one = salesman_route[city_one[tid]];
    int salesman_route_iminus_mod = salesman_route[(city_one[tid] - 1 + N) % N];
    int salesman_route_iplus_mod  = salesman_route[(city_one[tid] + 1) % N];
    
    // second city to swap
    int salesman_route_city_two = salesman_route[city_two[tid]];
    int salesman_route_kplus_mod  = salesman_route[city_two[tid] + 1 % N];
    int salesman_route_kminus_mod = salesman_route[(city_two[tid] - 1 + N) % N];
    
    // we should return this so we know the minimum route -S.
    delta = dist[salesman_route_iminus_mod * N + salesman_route_city_two] +
            dist[salesman_route_city_two * N + salesman_route_iplus_mod]  +
            dist[salesman_route_kminus_mod * N + salesman_route_city_one] +
            dist[salesman_route_city_one * N + salesman_route_kplus_mod]  -
            dist[salesman_route_iminus_mod * N + salesman_route_city_one] - 
            dist[salesman_route_city_one * N + salesman_route_iplus_mod] -
            dist[salesman_route_kminus_mod * N + salesman_route_city_two] - 
            dist[salesman_route_city_two * N + salesman_route_kplus_mod];
            
    if (delta < 0.0){
      flag[tid] = 1;
    } else {
      p = exp(-delta * b / T[0]);
      if (p > r[tid])
        flag[tid] = 1;
    }
 }
 
 
 
 /* Function to generate random numbers in interval
 
 input:
- min [unsigned integer(1)]
  - The minimum number to sample
- max [unsigned integer(1)]
  - The maximum number to sample
  
  Output: [unsigned integer(1)]
    - A randomly generated number between the range of min and max
    
  Desc:
  Taken from
  - http://stackoverflow.com/questions/2509679/how-to-generate-a-random-number-from-within-a-range
  
  
 */
 unsigned int rand_interval(unsigned int min, unsigned int max)
{
    int r;
    const unsigned int range = 1 + max - min;
    const unsigned int buckets = RAND_MAX / range;
    const unsigned int limit = buckets * range;

    /* Create equal size buckets all in a row, then fire randomly towards
     * the buckets until you land in one of them. All buckets are equally
     * likely. If you land off the end of the line of buckets, try again. */
    do
    {
        r = rand();
    } while (r >= limit);

    return min + (r / buckets);
}



 int main(){
 
     // start counters for cities
     unsigned int i, j, m;
     
     // city's x y coordinates
     struct coordinates {
         int x;
         int y;
     };
     
     struct coordinates location[N];
     
     unsigned int *salesman_route = (unsigned int *)malloc(N * sizeof(unsigned int));
     
     // just make one inital guess route, a simple linear path
     for (i = 0; i < N; i++)
         salesman_route[i] = i;
         
     // Set the starting and end points to be the same
     salesman_route[N-1] = salesman_route[0];
     
     // initialize the coordinates and sequence
     for(i = 0; i < N; i++){
         location[i].x = rand() % 1000;
         location[i].y = rand() % 1000;
     }
     
     // distance
     //float dist[N * N];
     float *dist = (float *)malloc(N * N * sizeof(float));
     
     for(i = 0; i < N; i++){
         for (j = 0; j < N; j++){
             // Calculate the euclidian distance between each city
             // use pow() here instead?
             dist[i * N + j] = (location[i].x - location[j].x) * (location[i].x - location[j].x) +
                               (location[j].y - location[j].y) * (location[i].y - location[j].y);
         }
     }
     // Calculate the original loss
     float original_loss = 0;
     for (i = 0; i < N - 1; i++){
         original_loss += dist[salesman_route[i] * N + salesman_route[i+1]];
     }
     printf("Original Loss is: %.6f \n", original_loss);
     // Keep the original loss for comparison pre/post algorithm
     float starting_loss = original_loss;
     float *dist_g, T = 999999999999, *T_g, *r_g;
     float *r_h = (float *)malloc(GRID_SIZE * sizeof(float));
     /*
     Defining device variables:
     city_swap_one_h/g: [integer(t_num)]
       - Host/Device memory for city one
     city_swap_two_h/g: [integer(t_num)]
       - Host/Device memory for city two
     flag_h/g: [integer(t_num)]
       - Host/Device memory for flag of accepted step
     salesman_route_g: [integer(N)]
       - Device memory for the salesmans route
     r_g:  [float(t_num)]
       - Device memory for the random number when deciding acceptance
     flag_h/g: [integer(t_num)]
       - host/device memory for acceptance vector
     original_loss_g: [integer(1)]
       - The device memory for the current loss function
     new_loss_h/g: [integer(t_num)]
       - The host/device memory for the proposal loss function
     */
     unsigned int *city_swap_one_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
     unsigned int *city_swap_two_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
     unsigned int *flag_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
     unsigned int *city_swap_one_g, *city_swap_two_g, *salesman_route_g, *flag_g;

     float new_loss_h = 0;
     
     hipError_t err = hipMalloc((void**)&city_swap_one_g, GRID_SIZE * sizeof(unsigned int));
     //printf("\n Cuda malloc city swap one: %s \n", hipGetErrorString(err));
     hipMalloc((void**)&city_swap_two_g, GRID_SIZE * sizeof(unsigned int));
     hipMalloc((void**)&dist_g, N * N * sizeof(float));
     hipMalloc((void**)&salesman_route_g, N * sizeof(unsigned int));
     hipMalloc((void**)&T_g, sizeof(float));
     hipMalloc((void**)&r_g, GRID_SIZE * sizeof(float));
     hipMalloc((void**)&flag_g, GRID_SIZE * sizeof(unsigned int));
     
     
     hipMemcpy(dist_g, dist, (N*N) * sizeof(float), hipMemcpyHostToDevice);
     // Beta is the decay rate
     float beta = 0.001;
     float a = T; 
     float f;
     
     while (T > 1){
         // Init parameters
         //printf("Current Temperature is: %.6f:", T);
         for(m = 0; m < GRID_SIZE; m++){
             // pick first city to swap
             city_swap_one_h[m] = rand_interval(1, N-2);
             // f defines how far the second city can be from the first
             f = exp(-a / T);
             j = (unsigned int)floor(1 + city_swap_one_h[m] * f); 
             // pick second city to swap
             city_swap_two_h[m] = (city_swap_one_h[m] + j) % N;
             // Check we are not at the first or last city for city two
             if (city_swap_two_h[m] == 0)
               city_swap_two_h[m] += 1;
             if (city_swap_two_h[m] == N - 1)
               city_swap_two_h[m] -= 1;
             r_h[m] = (float)rand() / (float)RAND_MAX ;
             
             //set our flags and new loss to 0
             flag_h[m] = 0;
          }
          // Copy memory from host to device
          err = hipMemcpy(city_swap_one_g, city_swap_one_h, GRID_SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);
          //printf("\n Cuda mem copy city swap one: %s \n", hipGetErrorString(err));
          hipMemcpy(city_swap_two_g, city_swap_two_h, GRID_SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);
          hipMemcpy(salesman_route_g, salesman_route, N * sizeof(unsigned int), hipMemcpyHostToDevice);
          hipMemcpy(T_g, &T, sizeof(float), hipMemcpyHostToDevice);
          hipMemcpy(r_g, r_h, GRID_SIZE * sizeof(float), hipMemcpyHostToDevice);
          hipMemcpy(flag_g, flag_h, GRID_SIZE* sizeof(unsigned int), hipMemcpyHostToDevice);
 
          // Number of thread blocks in grid
          dim3 blocksPerGrid(1,GRID_SIZE/t_num,1);
          dim3 threadsPerBlock(1,t_num,1);
    
          //static void tsp(int* city_one, int* city_two, float *dist, int *salesman_route,
          //                 float *T, float *r, int *flag){
          tsp<<<blocksPerGrid, threadsPerBlock, 0>>>(city_swap_one_g, city_swap_two_g,
                                                         dist_g, salesman_route_g,
                                                         T_g, r_g, flag_g);

          hipDeviceSynchronize();          
          hipMemcpy(flag_h, flag_g, GRID_SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
          /* 
          Here we check for a success
            The first proposal trip accepted becomes the new starting trip 
          */
          for (i = 0; i < GRID_SIZE; i++){
              if (flag_h[i] == 0){
              //printf("Original Loss: %.6f \n", original_loss);
              //printf("Proposed Loss: %.6f \n", new_loss_h[i]);
                  continue;
              } else {
                  // switch the two cities that led to an accepted proposal
                  unsigned int tmp = salesman_route[city_swap_one_h[i]];
                  salesman_route[city_swap_one_h[i]] = salesman_route[city_swap_two_h[i]];
                  salesman_route[city_swap_two_h[i]] = tmp;
                  for (i = 0; i < N - 1; i++){
                    new_loss_h += dist[salesman_route[i] * N + salesman_route[i+1]];
                  }
                  // set old loss function to new
                  original_loss = new_loss_h;
                  //decrease temp
                  T -= T*beta;
                  //if (T < 300){
                    printf(" Current Temperature is %.6f \n", T);
                    printf("\n Current Loss is: %.6f \n", original_loss);
                  //}
                  /*
                  printf("Best found trip so far\n");
                  for (j = 0; j < N; j++){
                     printf("%d ", salesman_route[j]);
                  }
                  */
                  break;
              }
          }
     }
     printf("The starting loss was %.6f and the final loss was %.6f \n", starting_loss, original_loss);
     /*
     printf("\n Final Route:\n");
     for (i = 0; i < N; i++)
       printf("%d ",salesman_route[i]);
     */    
     hipFree(city_swap_one_g);
     hipFree(city_swap_two_g);
     hipFree(dist_g);
     hipFree(salesman_route_g);
     hipFree(T_g);
     hipFree(r_g);
     hipFree(flag_g);
     free(dist);
     free(salesman_route);
     free(city_swap_one_h);
     free(city_swap_two_h);
     free(flag_h);
     return 0;
}
             
         
         
         

