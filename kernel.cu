#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>
#include <limits.h>
#include <assert.h>
#include <iostream>
#include <fstream>


#include "utils.h"
#include "tsp_solve.h"


#define t_num 1024
#define GRID_SIZE 9216

/*
For more samples define GRID_SIZE as a multiple of t_num such as 512000, 2048000, or the (max - 1024) grid size 2147482623
Some compliation options that can speed things up
--use_fast_math
--optimize=5
--gpu-architecture=compute_35
I use something like
NOTE: You need to use the -lcurand flag to compile.
nvcc --optimize=5 --use_fast_math -arch=compute_35 kernel.cu -o tsp_cuda -lcurand
*/

int main(){

	const char *tsp_name = "dsj1000.tsp";
	read_tsp(tsp_name);
	unsigned int N = meta->dim, *N_g;
	// start counters for cities
	unsigned int i;

	coordinates *location_g;

	/* For checking the coordinates
	for (i = 0; i < N; i++)
	printf("Location x: %0.6f, location y: %0.6f \n", location[i].x, location[i].y);
	exit(0);
	*/
	unsigned int *salesman_route = (unsigned int *)malloc((N + 1) * sizeof(unsigned int));

	// just make one inital guess route, a simple linear path
	for (i = 0; i <= N; i++)
		salesman_route[i] = i;

	// Set the starting and end points to be the same
	salesman_route[N] = salesman_route[0];

	/*     don't need it when importing data from files
	// initialize the coordinates and sequence
	for(i = 0; i < N; i++){
	location[i].x = rand() % 1000;
	location[i].y = rand() % 1000;
	}
	*/



	// Calculate the original loss
	float original_loss = 0;
	for (i = 0; i < N; i++){
		original_loss += (location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
			(location[salesman_route[i]].x - location[salesman_route[i + 1]].x) +
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y);
	}
	printf("Original Loss is:  %0.6f \n", original_loss);
	// Keep the original loss for comparison pre/post algorithm
	// SET THE LOSS HERE
	float T[1], *T_g;
	T[0] = 30;
	/*
	Defining device variables:
	city_swap_one_h/g: [integer(t_num)]
	- Host/Device memory for city one
	city_swap_two_h/g: [integer(t_num)]
	- Host/Device memory for city two
	flag_h/g: [integer(t_num)]
	- Host/Device memory for flag of accepted step
	salesman_route_g: [integer(N)]
	- Device memory for the salesmans route
	flag_h/g: [integer(t_num)]
	- host/device memory for acceptance vector
	original_loss_g: [integer(1)]
	- The device memory for the current loss function
	(DEPRECATED)new_loss_h/g: [integer(t_num)]
	- The host/device memory for the proposal loss function
	*/
	unsigned int *city_swap_one_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
	unsigned int *city_swap_two_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
	unsigned int *flag_h = (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
	unsigned int *salesman_route_g, *salesman_route_2g, *flag_g, *city_swap_one_g, *city_swap_two_g;
	unsigned int global_flag_h = 0, *global_flag_g;

	hipMalloc((void**)&city_swap_one_g, GRID_SIZE * sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&city_swap_two_g, GRID_SIZE * sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&location_g, N * sizeof(coordinates));
	cudaCheckError();
	hipMalloc((void**)&salesman_route_g, (N + 1) * sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&salesman_route_2g, (N + 1) * sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&T_g, sizeof(float));
	cudaCheckError();
	hipMalloc((void**)&flag_g, GRID_SIZE * sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&global_flag_g, sizeof(unsigned int));
	cudaCheckError();
	hipMalloc((void**)&N_g, sizeof(unsigned int));
	cudaCheckError();


	hipMemcpy(location_g, location, N * sizeof(coordinates), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(salesman_route_g, salesman_route, (N + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(salesman_route_2g, salesman_route, (N + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(global_flag_g, &global_flag_h, sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(N_g, &N, sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaCheckError();
	// Beta is the decay rate
	//float beta = 0.0001;
	// We are going to try some stuff for temp from this adaptive simulated annealing paper
	// https://arxiv.org/pdf/cs/0001018.pdf

	// Number of thread blocks in grid
	// X is for the sampling, y is for manipulating the salesman's route
	dim3 blocksPerSampleGrid(GRID_SIZE / t_num, 1, 1);
	dim3 blocksPerTripGrid((N / t_num) + 1, 1, 1);
	dim3 threadsPerBlock(t_num, 1, 1);

	// Trying out random gen in cuda
	hiprandState_t* states;

	/* allocate space on the GPU for the random states */
	hipMalloc((void**)&states, GRID_SIZE * sizeof(hiprandState_t));
	init <<<blocksPerSampleGrid, threadsPerBlock, 0 >>>(time(0), states);

	//time counter
	time_t t_start, t_end;
	t_start = time(NULL);

	while (T[0] > 1)
	{
		// Copy memory from host to device
		hipMemcpy(T_g, T, sizeof(float), hipMemcpyHostToDevice);
		cudaCheckError();
		i = 1;

		while (i<500){

			
			tspSwap <<<blocksPerSampleGrid, threadsPerBlock, 0 >>>(city_swap_one_g, city_swap_two_g,
				                                                   location_g, salesman_route_g,
				                                                   T_g, global_flag_g, N_g,
				                                                   states);
			cudaCheckError(); 
			tspSwapUpdate <<<blocksPerSampleGrid, threadsPerBlock, 0 >>>(city_swap_one_g, city_swap_two_g,
				                                                         salesman_route_g, global_flag_g);
			cudaCheckError();
			
			tspSwap2 << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				                                                      location_g, salesman_route_g,
				                                                      T_g, global_flag_g, N_g,
				                                                      states);
			cudaCheckError();
			
			tspSwapUpdate << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				salesman_route_g, global_flag_g);
			cudaCheckError();
			/*

			tspInsertion <<<blocksPerSampleGrid, threadsPerBlock, 0 >>>(city_swap_one_g, city_swap_two_g,
			                                                            location_g, salesman_route_g,
			                                                            T_g, global_flag_g, N_g,
			                                                            states); 
			cudaCheckError();
			
			tspInsertionUpdateTrip << <blocksPerTripGrid, threadsPerBlock, 0 >> >(salesman_route_g, salesman_route_2g, N_g);
			cudaCheckError();
			
			tspInsertionUpdate2 <<<blocksPerTripGrid, threadsPerBlock, 0 >>>(city_swap_one_g, city_swap_two_g,
				                                                             salesman_route_g, salesman_route_2g,
				                                                             global_flag_g);
            cudaCheckError();
            
		    tspInsertionUpdateTrip << <blocksPerTripGrid, threadsPerBlock, 0 >> >(salesman_route_g, salesman_route_2g, N_g);
			cudaCheckError();

			tspInsertion2 << <blocksPerSampleGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				                                                           location_g, salesman_route_g,
				                                                           T_g, global_flag_g, N_g,
				                                                           states);
			cudaCheckError();

			tspInsertionUpdate2 << <blocksPerTripGrid, threadsPerBlock, 0 >> >(city_swap_one_g, city_swap_two_g,
				                                                               salesman_route_g, salesman_route_2g,
				                                                               global_flag_g);
			cudaCheckError();
			
			tspInsertionUpdateTrip << <blocksPerTripGrid, threadsPerBlock, 0 >> >(salesman_route_g, salesman_route_2g, N_g);
			cudaCheckError();
			*/
	//		iter += 1.00f;
	//		T = T_start / log(iter);
	//		if ((long int)iter % 50000 == 0)
	//			printf("Iter: %ld  Temperature is %.6f\n", (long int)iter, T);
			//T = 1;
			i++;
		}
		hipMemcpy(salesman_route, salesman_route_g, (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
		cudaCheckError();
		float optimized_loss = 0;
		for (i = 0; i < N; i++){
			optimized_loss += (location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
				(location[salesman_route[i]].x - location[salesman_route[i + 1]].x) +
				(location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
				(location[salesman_route[i]].y - location[salesman_route[i + 1]].y);
		}
		
		T[0] = T[0] * 0.999;
		printf("T[0]: %f Optimized Loss is: %.6f  \n",T[0], optimized_loss);
	}
	//print time spent
	t_end = time(NULL);
	printf("time = %f\n", difftime(t_end, t_start));

	hipMemcpy(salesman_route, salesman_route_g, (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	cudaCheckError();
	float optimized_loss = 0;
	for (i = 0; i < N; i++){
		optimized_loss += (location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
			(location[salesman_route[i]].x - location[salesman_route[i + 1]].x) +
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
			(location[salesman_route[i]].y - location[salesman_route[i + 1]].y);
	} 
	printf("Optimized Loss is: %.6f \n", optimized_loss);

	// Write the best trip to CSV
	FILE *best_trip;
	const char *filename = "mona_lisa_best_trip.csv";
	best_trip = fopen(filename, "w+");
	fprintf(best_trip, "location,coordinate_x,coordinate_y\n");
	for (i = 0; i < N + 1; i++){
		fprintf(best_trip, "%d,%.6f,%.6f\n",
			salesman_route[i],
			location[salesman_route[i]].x,
			location[salesman_route[i]].y);
	}
	fclose(best_trip);

	/*
	printf("\n Final Route:\n");
	for (i = 0; i < N; i++)
	printf("%d ",salesman_route[i]);
	*/
	hipFree(location_g);
	cudaCheckError();
	hipFree(salesman_route_g);
	cudaCheckError();
	hipFree(salesman_route_2g);
	cudaCheckError();
	hipFree(T_g);
	cudaCheckError();
	hipFree(flag_g);
	cudaCheckError();
	free(salesman_route);
	free(city_swap_one_h);
	free(city_swap_two_h);
	free(flag_h);
	free(location);
	getchar();
	return 0;
}

